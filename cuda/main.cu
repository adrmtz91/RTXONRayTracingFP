#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"
#include "color.h"
#include "rtweekend.h"
#include <hip/hip_runtime.h>

// ############################################################################
//                          KERNEL CODE
// ############################################################################

// Device function for color calculation
__device__ color ray_color(const ray& r, hittable** world) {
    hit_record rec;
    if ((*world)->hit(r, interval(0.0, inf), rec)) {
        vec3 N = rec.normal;
        return 0.5 * color(N.x() + 1, N.y() + 1, N.z() + 1);
    }
    vec3 unit_direction = unit_vector(r.direction());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

// Rendering the scene
__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pixel, camera cam, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    color pixel_color(0, 0, 0);
    for (int s = 0; s < samples_per_pixel; ++s) {
        auto u = double(i + random_double(&local_rand_state)) / (max_x - 1);
        auto v = double(j + random_double(&local_rand_state)) / (max_y - 1);
        ray r = cam.get_ray(u, v, &local_rand_state);
        pixel_color += ray_color(r, world);
    }
    rand_state[pixel_index] = local_rand_state;
    fb[pixel_index] = pixel_color / samples_per_pixel;
}

// Creating the world
__global__ void create_world(hittable** d_list, hittable** d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(point3(0, 0, -1), 0.5);
        d_list[1] = new sphere(point3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_list, 2);
    }
}

// Freeing the world
__global__ void free_world(hittable** d_list, hittable** d_world) {
    delete ((sphere*)d_list[0]);
    delete ((sphere*)d_list[1]);
    delete ((hittable_list*)*d_world);
}

// Initializing random states
__global__ void init_rand_state_kernel(hiprandState* rand_state, unsigned long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &rand_state[idx]);
}

// ############################################################################
//                          MAIN CODE
// ############################################################################

int main() {
    // Initialize host variables ----------------------------------------------
    // Image
    const int image_width = 400;
    const int image_height = static_cast<int>(image_width / 16.0 * 9.0);
    const int samples_per_pixel = 100;
    
    // Allocate device variables ----------------------------------------------

    // Allocate FB
    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);
    vec3* fb;
    hipMalloc((void**)&fb, fb_size);

    // Random state
    hiprandState* d_rand_state;
    hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState));

    // Initialize frame buffer
    hipMemset(fb, 0, fb_size);

    // World
    hittable** d_list;
    hipMalloc((void**)&d_list, 2 * sizeof(hittable*));
    hittable** d_world;
    hipMalloc((void**)&d_world, sizeof(hittable*));
    create_world<<<1, 1>>>(d_list, d_world);
    hipDeviceSynchronize();
    
    // Launch kernel ----------------------------------------------------------

    // Initialize random states
    init_rand_state_kernel<<<(num_pixels + 255) / 256, 256>>>(d_rand_state, time(0));
    hipDeviceSynchronize();

    // Camera
    camera cam;

    // Timing variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);
    
    // Initialize thread block and kernel grid dimensions ---------------------
    // Render buffer
    dim3 blocks((image_width + 15) / 16, (image_height + 15) / 16);
    dim3 threads(16, 16);

    // Invoke CUDA kernel -----------------------------------------------------

    render<<<blocks, threads>>>(fb, image_width, image_height, samples_per_pixel, cam, d_world, d_rand_state);
    hipDeviceSynchronize();

    // Allocate host memory for the frame buffer
    vec3* host_fb = new vec3[num_pixels];

    // Copy device variables from host ----------------------------------------

    hipMemcpy(host_fb, fb, fb_size, hipMemcpyDeviceToHost);

    // Output FB as image
    std::ofstream output_file("output_cuda.ppm");
    output_file << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; --j) {
        std::clog << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; ++i) {
            int pixel_index = j * image_width + i;
            write_color(output_file, host_fb[pixel_index]);
        }
    }
    output_file.close();

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "\nRender Time (CUDA): " << milliseconds << " milliseconds\n";

    // Free memory ------------------------------------------------------------
    delete[] host_fb;
    free_world<<<1, 1>>>(d_list, d_world);
    hipDeviceSynchronize();
    hipFree(d_world);
    hipFree(d_list);
    hipFree(fb);
    hipFree(d_rand_state);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
